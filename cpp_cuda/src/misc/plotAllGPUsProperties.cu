#include"gpuDeviceProperties.h"


int get_AllDevicesProperties(hipDeviceProp_t* dev_prop)
{
	int dev_count; hipGetDeviceCount(&dev_count);
	for (int deviceIndex = 0; deviceIndex < dev_count; deviceIndex++) hipGetDeviceProperties(&dev_prop[deviceIndex], deviceIndex);
	
	return dev_count;
}


void plotAllGPUsProperties()
{
  int dev_count; hipGetDeviceCount(&dev_count);
  hipDeviceProp_t dev_prop[dev_count];
  for (int deviceIndex = 0; deviceIndex < dev_count; deviceIndex++) hipGetDeviceProperties(&dev_prop[deviceIndex], deviceIndex);


  std::cout << "\n----------------------------\n| Number of GPU devices: " << dev_count << " |\n----------------------------\n" << std::endl;


  for (int deviceIndex = 0; deviceIndex < dev_count; deviceIndex++) 
  {
     std::cout << "\nProperties of GPU device nr. " << deviceIndex << "\n------------------------------" << std::endl;
     plotGPUProperties(dev_prop[deviceIndex]);
  }

}
